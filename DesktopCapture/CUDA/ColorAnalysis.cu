#include "hip/hip_runtime.h"
#include "ColorAnalysis.h"
#include "Logger.h"
#include <hip/hip_runtime.h>
#include <>
#include <cinttypes>
#include <hiprand.h>

__device__ unsigned int output[4] = { 0, 0, 0, 0 };

__global__
void calculateMeanColor(uint32_t *screen, int width, int height, size_t pitch)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return;

	uint32_t val = screen[y * width + x];
	atomicAdd(output, val & 0xFF);
	atomicAdd(output + 1, (val >> 8) & 0xFF);
	atomicAdd(output + 2, (val >> 16) & 0xFF);
}


namespace CudaUtils
{
	RgbColor getMeanColor(hipGraphicsResource* texture, void* buf, int width, int height, size_t pitch)
	{
		hipArray* cuArray;
		hipError_t status = hipGraphicsSubResourceGetMappedArray(&cuArray, texture, 0, 0);
		status = hipMemcpy2DFromArray(buf, pitch, cuArray, 0, 0, pitch, height, hipMemcpyDeviceToDevice);

		unsigned int result[4] = { 0,0,0,0 };
		status = hipMemcpyToSymbol(HIP_SYMBOL(output), result, sizeof(result), 0, hipMemcpyHostToDevice);

		// kick off the kernel and send the staging buffer cudaLinearMemory as an argument to allow the kernel to write to it
		dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
		dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
		calculateMeanColor<<<Dg, Db>>>((uint32_t*) buf, width, height, pitch);

		hipError_t error = hipDeviceSynchronize();
		if (error != hipSuccess)
		{
			LOGSEVERE("cuda mean color failed to launch wth error %d\n", error);
			return { 0,0,0 };
		}
		status = hipMemcpyFromSymbol(result, HIP_SYMBOL(output), sizeof(result), 0, hipMemcpyDeviceToHost);
		int pixelCount = width * height;
		RgbColor c = { static_cast<uint8_t>(result[2] / pixelCount), static_cast<uint8_t>(result[1] / pixelCount), static_cast<uint8_t>(result[0] / pixelCount) };
		return c;
	}
}
