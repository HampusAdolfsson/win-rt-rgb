#include "hip/hip_runtime.h"
#include "ColorAnalysis.h"
#include "Logger.h"
#include <hip/hip_runtime.h>
#include <>
#include <cinttypes>
#include <hiprand.h>

__device__ int output[4] = { 0, 0, 0, 0 };

__global__
void calculateMeanColor(hipTextureObject_t screen, int width, int height, size_t pitch)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float u = x / (float)width;
	float v = y / (float)height;
	if (x < width && y < height) {
		int val = tex2D<int>(screen, u, v);
		//int val = surface[y * pitch + x];
		int channel = x % 4;
		printf("%f,%f,%.8X\n", u, v, val);
		//surface[y * pitch + x] = 0xff00ff;
		atomicAdd(output + channel, val);
	}
}
__global__
void calculateMeanColor2(int *screen, int width, int height, size_t pitch)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float u = x / (float)width;
	float v = y / (float)height;
	if (x < width && y < height) {
		int val = screen[y * pitch + x];
		int channel = x % 4;
		printf("%f,%f,%.8X\n", u, v, val);
		screen[y * pitch + x] = 0xff00ff;
		atomicAdd(output + channel, val);
	}
}


namespace CudaUtils
{
	//Color getMeanColor(hipGraphicsResource* texture, void* buf, int width, int height, size_t pitch)
	//{
	//	hipArray* cuArray;
	//	hipError_t status = hipGraphicsSubResourceGetMappedArray(&cuArray, texture, 0, 0);
	//	status = hipMemcpy2DFromArray(buf, pitch, cuArray, 0, 0, pitch, height, hipMemcpyDeviceToDevice);

	//	struct hipResourceDesc resDesc;
	//	memset(&resDesc, 0, sizeof(resDesc));
	//	resDesc.res.array.array = cuArray;
	//	resDesc.resType = hipResourceTypeArray;

	//	struct hipTextureDesc texDesc;
	//	memset(&texDesc, 0, sizeof(texDesc));
	//	texDesc.addressMode[0] = hipAddressModeClamp;
	//	texDesc.addressMode[1] = hipAddressModeClamp;
	//	texDesc.filterMode = hipFilterModePoint;
	//	texDesc.readMode = hipReadModeElementType;
	//	texDesc.normalizedCoords = 1;

	//	hipTextureObject_t texObj = 0;
	//	status = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

	//	// kick off the kernel and send the staging buffer cudaLinearMemory as an argument to allow the kernel to write to it
	//	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	//	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
	//	calculateMeanColor2<<<Dg, Db>>>((int*)buf, width, height, pitch);

	//	hipError_t error = hipDeviceSynchronize();
	//	if (error != hipSuccess)
	//	{
	//		LOGSEVERE("cuda mean color failed to launch wth error %d\n", error);
	//		return { 0,0,0 };
	//	}
	//	int result[4];
	//	status = hipMemcpyFromSymbol(result, HIP_SYMBOL(output), sizeof(result), 0, hipMemcpyDeviceToHost);
	//	int pixelCount = width * height;
	//	Color c = { static_cast<uint8_t>(result[2] / pixelCount), static_cast<uint8_t>(result[1] / pixelCount), static_cast<uint8_t>(result[0] / pixelCount) };

	//	// then we want to copy cudaLinearMemory to the D3D texture, via its mapped form : hipArray
	//	//status = hipMemcpy2DToArray(
	//	//	cuArray, // dst array
	//	//	0, 0,    // offset
	//	//	buf, pitch,       // src
	//	//	width * 4 * sizeof(float), height, // extent
	//	//	hipMemcpyDeviceToDevice); // kind
	//	hipDestroyTextureObject(texObj);
	//	return c;
	//}

	Color getMeanColor(hipGraphicsResource* texture, void* buf, int width, int height, size_t pitch) {
		hipArray* cuArray;
		hipError_t status = hipGraphicsSubResourceGetMappedArray(&cuArray, texture, 0, 0);
		status = hipMemcpy2DFromArray(buf, pitch, cuArray, 0, 0, pitch, height, hipMemcpyDeviceToDevice);
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
		//cuda Array
		hipArray* d_cuArr;
		hipMallocArray(&d_cuArr, &channelDesc, width, height);

		dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
		dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
		calculateMeanColor2<< <Dg, Db >> > ((int*)buf, 128, 128, 128);
		hipError_t error = hipDeviceSynchronize();
		if (error != hipSuccess)
		{
			LOGSEVERE("cuda mean color failed to launch wth error %d\n", error);
			return { 0,0,0 };
		}
		status = hipMemcpy2DToArray(d_cuArr, 0, 0, buf, pitch, width * sizeof(float), height, hipMemcpyDeviceToDevice);
			return { 0,0,0 };
	}
}
