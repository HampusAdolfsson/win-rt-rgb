#include "hip/hip_runtime.h"
#include "CudaKernels.h"
#include "Logger.h"
#include <hip/hip_runtime.h>
#include <>
#include <cinttypes>
#include <hiprand.h>
#include <assert.h>

__global__
void calculateMeanColorKernel(uint8_t* screen, int width, int height, size_t pitch, unsigned int* output, int outputWidth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return;
	unsigned int* destination = output + 3 * (x / outputWidth);

	uint32_t* pixel = (uint32_t*)(screen + y * pitch + x * sizeof(uint32_t));
	uint32_t val = *pixel;
	// TODO: make this more efficient
	atomicAdd(destination, val & 0xFF);
	atomicAdd(destination + 1, (val >> 8) & 0xFF);
	atomicAdd(destination + 2, (val >> 16) & 0xFF);
}

__global__
void averageAndAdjustColorsKernel(unsigned int* channels, int pixelsPerChannel, uint8_t* colorOutputs, int nOutputs, int outputPitch)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x > nOutputs) return;
	colorOutputs[outputPitch*x] = channels[3*x+2] / pixelsPerChannel;
	colorOutputs[outputPitch*x + 1] = channels[3*x+1] / pixelsPerChannel;
	colorOutputs[outputPitch*x + 2] = channels[3*x] / pixelsPerChannel;
	// TODO: do some color adjustments
}

__device__ unsigned int outputt[4] = { 0, 0, 0, 0 };

namespace CudaKernels
{
	void calculateMeanColor(uint8_t* pixels, int width, int height, size_t pitch, unsigned int* outputChannels, int outputWidth)
	{
		dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
		dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
		calculateMeanColorKernel<<<Dg, Db>>>(pixels, width, height, pitch, outputChannels, outputWidth);
	}

	void averageAndAdjustColors(unsigned int* channels, int pixelsPerChannel, uint8_t* colorOutputs, int outputSize, int outputPitch)
	{
		size_t blocksize = 128;
		size_t gridsize = (outputSize + blocksize - 1) / blocksize;
		averageAndAdjustColorsKernel<<<gridsize, blocksize>>>(channels, pixelsPerChannel, colorOutputs, outputSize, outputPitch);
	}

	void getMeanColor(hipGraphicsResource* texture, void* buf, size_t pitch, int width, int height, Rect activeRegion, RgbColor* output, int outputSize, void* intermediaryBuffer, void* cudaOutput)
	{
		assert(texture && buf);


		// status = hipMemcpyToSymbol(HIP_SYMBOL(output), result, sizeof(result), 0, hipMemcpyHostToDevice);
		// hipError_t status = hipMemset(intermediaryBuffer, 0, 3*sizeof(int)*outputSize);

		dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
		dim3 Dg = dim3((activeRegion.width + Db.x - 1) / Db.x, (activeRegion.height + Db.y - 1) / Db.y);
		calculateMeanColorKernel<<<Dg, Db>>>((uint8_t*) buf, activeRegion.width, activeRegion.height, pitch, (unsigned int*) intermediaryBuffer, activeRegion.width / outputSize);

		size_t blocksize = 128;
		size_t gridsize = (outputSize + blocksize - 1) / blocksize;
		averageAndAdjustColorsKernel<<<gridsize, blocksize>>>((unsigned int*)intermediaryBuffer, activeRegion.height * (activeRegion.width / outputSize), (uint8_t*)cudaOutput, outputSize, sizeof(RgbColor));

		hipError_t status = hipDeviceSynchronize();
		if (status != hipSuccess)
		{
			LOGSEVERE("cuda mean color failed to launch wth error %d\n", status);
			return;
		}
		status = hipMemcpy(output, cudaOutput, sizeof(RgbColor) * outputSize, hipMemcpyDeviceToHost);
	}
}
